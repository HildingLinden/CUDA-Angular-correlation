#include "hip/hip_runtime.h"
#include <stdio.h>

__global__
void saxpy(int n, float a, float *x, float *y) {
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i < n) y[i] = a*x[i] + y[i];
}

int main(void) {
	// Context creation
	//hipFree(0);

	int deviceCount;
	hipGetDeviceCount(&deviceCount);
	for (int i = 0; i < deviceCount; i++) {
		hipDeviceProp_t props;
		hipGetDeviceProperties(&props, i);
		printf("Device nr.: %d\n", i);
		printf("  Device name: %s\n", props.name);
		printf("  Memory clock rate: (MHz) %lf\n", props.memoryClockRate/1000.0);
		printf("  Memory bus width (bits): %d\n", props.memoryBusWidth);
		printf("  Peak memory bandwith (GB/s): %f\n", 2.0*props.memoryClockRate*(props.memoryBusWidth/8)/1.0e6);
		printf("  Compute capability: %d.%d\n\n", props.major, props.minor);
	}

	int n = 1 << 20;
	float *x, *y, *d_x, *d_y;
	// Using pinned host memory to speed up transfer
	hipHostMalloc((void **)&x, n*sizeof(float));
	hipHostMalloc((void **)&y, n*sizeof(float));

	hipMalloc((void **)&d_x, n*sizeof(float));
	hipMalloc((void **)&d_y, n*sizeof(float));

	for (int i = 0; i < n; i++) {
		x[i] = 1.0f;
		y[i] = 2.0f;
	}

	hipMemcpy(d_x, x, n*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_y, y, n*sizeof(float), hipMemcpyHostToDevice);

	saxpy<<<(n+255)/256, 256>>>(n, 2.0f, d_x, d_y);

	hipError_t errSync = hipGetLastError();
	hipError_t errAsync = hipDeviceSynchronize();

	if (errSync != hipSuccess) {
		printf("Sync kernel error: %s\n", hipGetErrorString(errSync));
	}
	if (errAsync != hipSuccess) {
		printf("Async kernel error: %s\n", hipGetErrorString(errAsync));
	}

	hipMemcpy(y, d_y, n*sizeof(float), hipMemcpyDeviceToHost);

	float maxError = 0.0f;
	for (int i = 0; i < n; ++i) {
		maxError = max(maxError, abs(y[i]-4.0f));
	}
		printf("Max error: %f\n", maxError);

	hipFree(d_x);
	hipFree(d_y);
	hipHostFree(x);
	hipHostFree(y);
}