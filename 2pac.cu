#include "hip/hip_runtime.h"

#include <stdio.h>
#include <fstream>
#include <iostream>
#include <math.h>

const int ROWSPERTHREAD = 512;

// 2-point angular correlation
__global__
void DR_kernel(int nCols, int nRows, float *d, double *r, unsigned int *DR) {
	// The thread id on the x-axis and y-axis
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * ROWSPERTHREAD;

	// blockIdx.y * ROWSPERTHREAD

	if (x < nCols) {
		__shared__ unsigned int hist[720];
		//__shared__ double sharedR[ROWSPERTHREAD * 2 + 1];

		// Right ascension of R
		//sharedR[threadIdx.x * 2] = r[(y + threadIdx.x) * 2];
		// Declination of R
		//sharedR[threadIdx.x * 2 + 1] = r[(y + threadIdx.x) * 2 + 1];

		// Right ascension and declination for the current element
		float asc1 = d[x * 2];
		float dec1 = d[x * 2 + 1];

		float decimalResult;
		// n-y is the distance to the domain edge
		int nElements = min(nRows-y, ROWSPERTHREAD);

		//__syncthreads();

		for (int j = 0; j < nElements; j++) {
			double asc2 = r[y + j * 2];
			double dec2 = r[y + j * 2 + 1];

			if (fabs(asc1-asc2) > 0.0001f && fabs(dec1-dec2) > 0.0001f) {
				decimalResult = acos(sinf(dec1) * sin((float)dec2) + cos(dec1) * cos((float)dec2) * cos(asc1-(float)asc2));
				int resultIndex = floor(decimalResult/0.25);
				atomicAdd(&hist[resultIndex], 1);
			}
		}

		__syncthreads();

		if (threadIdx.x == 0) {
			for (int i = 0; i < 720; i++) {
				atomicAdd(&DR[i], hist[i]);
			}
		}
	}
}

int main(void) {
	// Info about the GPU
	int deviceCount;
	hipGetDeviceCount(&deviceCount);
	for (int i = 0; i < deviceCount; i++) {
		hipDeviceProp_t props;
		hipGetDeviceProperties(&props, i);
		printf("\nDevice nr.: %d\n", i);
		printf("  Device name: %s\n", props.name);
		printf("  Memory clock rate: (MHz) %lf\n", props.memoryClockRate/1000.0);
		printf("  Memory bus width (bits): %d\n", props.memoryBusWidth);
		printf("  Peak memory bandwith (GB/s): %f\n", 2.0*props.memoryClockRate*(props.memoryBusWidth/8)/1.0e6);
		printf("  Compute capability: %d.%d\n", props.major, props.minor);
		printf("  Shared memory per block: %zd\n", props.sharedMemPerBlock);
		printf("  Multiprocessor count: %d\n\n", props.multiProcessorCount);
	}

	//hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte);


	// Read real data file
	std::ifstream infileD("data_100k_arcmin.txt");

	// Get amount of coordinate pairs
	int nCoordinatePairsD;
	infileD >> nCoordinatePairsD;
	printf("Found %d coordinate pairs in data\n", nCoordinatePairsD);

	// Allocate memory for real data on host
	float *h_D = (float *)malloc(nCoordinatePairsD * 2 * sizeof(float));

	// Read synthetic data file
	std::ifstream infileR("flat_100k_arcmin.txt");

	// Get amount of coordinate pairs
	int nCoordinatePairsR;
	infileR >> nCoordinatePairsR;
	printf("Found %d coordinate pairs in flat\n", nCoordinatePairsR);

	// Allocate memory for synthetic data on host
	double *h_R = (double *)malloc(nCoordinatePairsR * 2 * sizeof(double));

	if (h_D == NULL || h_R == NULL) printf("Allocating memory on host failed");


	int index = 0;

	// Initialize data
	float ascD, decD;
	while (infileD >> ascD >> decD) {
		if (index < nCoordinatePairsD * 2) {
			h_D[index++] = ascD;
			h_D[index++] = decD;
		} else {
			printf("Number of coordinate pairs given in file does not match the actual amount in data\n");
			exit(1);
		}
	}

	// Initialize synthetic
	double ascR, decR;
	index = 0;
	while (infileR >> ascR >> decR) {
		if (index < nCoordinatePairsR * 2) {
			h_R[index++] = ascR;
			h_R[index++] = decR;
		} else {
			printf("Number of coordinate pairs given in file does not match the actual amount in flat\n");
			exit(1);
		}
	}


	// Allocating and copying the input data to device
	float *d_D;
	double *d_R;

	hipMalloc((void **)&d_D, nCoordinatePairsD * 2 * sizeof(float));
	hipMalloc((void **)&d_R, nCoordinatePairsR * 2 * sizeof(double));

	hipMemcpy(d_D, h_D, nCoordinatePairsD * 2 * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_R, h_R, nCoordinatePairsR * 2 * sizeof(double), hipMemcpyHostToDevice);


	// Allocating and zero-initializing the result arrays on host
	unsigned int *h_DD, *h_DR, *h_RR;
	h_DD = (unsigned int *)calloc(720, sizeof(unsigned int));
	h_DR = (unsigned int *)calloc(720, sizeof(unsigned int));
	h_RR = (unsigned int *)calloc(720, sizeof(unsigned int));

	// Allocating the result arrays on device
	unsigned int *d_DD, *d_DR, *d_RR;
	hipMalloc((void **)&d_DD, 720 * sizeof(unsigned int));
	hipMalloc((void **)&d_DR, 720 * sizeof(unsigned int));
	hipMalloc((void **)&d_RR, 720 * sizeof(unsigned int));

	// Copying the zero-initialized arrays to the GPU
	hipMemcpy(d_DD, h_DD, 720 * sizeof(unsigned int), hipMemcpyHostToDevice);
	hipMemcpy(d_DR, h_DR, 720 * sizeof(unsigned int), hipMemcpyHostToDevice);
	hipMemcpy(d_RR, h_RR, 720 * sizeof(unsigned int), hipMemcpyHostToDevice);

	// Calculating sizes and launching kernel
	int blockSize = 256;
	int gridSizeX = (nCoordinatePairsD/2 + blockSize - 1) / blockSize;
	int gridSizeY = (nCoordinatePairsR/2 + blockSize - 1) / blockSize;
	dim3 gridSize2D(gridSizeX, gridSizeY);

	DR_kernel<<<gridSize2D, blockSize>>>(nCoordinatePairsD, nCoordinatePairsR, d_D, d_R, d_DR);

	// Checking for errors
	hipError_t errSync = hipGetLastError();
	hipError_t errAsync = hipDeviceSynchronize();

	if (errSync != hipSuccess) {
		printf("Sync kernel error: %s\n", hipGetErrorString(errSync));
	}
	if (errAsync != hipSuccess) {
		printf("Async kernel error: %s\n", hipGetErrorString(errAsync));
	}

	// Copying the result from device to host
	// hipMemcpy has an implicit barrier
	hipMemcpy(h_DD, d_DD, 720 * sizeof(unsigned int), hipMemcpyDeviceToHost);
	hipMemcpy(h_DR, d_DR, 720 * sizeof(unsigned int), hipMemcpyDeviceToHost);
	hipMemcpy(h_RR, d_RR, 720 * sizeof(unsigned int), hipMemcpyDeviceToHost);

	for (int i = 0; i < 20; i++) {
		printf("%d: %u\n", i, h_DR[i]);
	}

	// // Computing the difference
	// double *result;
	// result = (double *)malloc(sizeof(double) * 720);
	// printf("\nResult:\n");
	// for (int i = 0; i < 720; i++) {
	// 	if(h_RR[i] == 0) {
	// 		result[i] = 0.0;
	// 	} else {
	// 		result[i] = (h_DD[i] - 2 * h_DR[i] + h_RR[i]) / (double)h_RR[i];
	// 	}
	// 	printf("%d: %lf\n", i, result[i]);
	// }

	hipFree(d_D);
	hipFree(d_R);
	hipFree(d_DD);
	hipFree(d_DR);
	hipFree(d_RR);
	free(h_D);
	free(h_R);
	free(h_DD);
	free(h_DR);
	free(h_RR);
}
