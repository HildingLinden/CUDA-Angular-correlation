#include "hip/hip_runtime.h"
#include <stdio.h>

__global__
void saxpy(int n, float a, float *x, float *y) {
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i < n) y[i] = a*x[i] + y[i];
}

int main(void) {
	// Context creation
	//hipFree(0);

	int deviceCount;
	hipGetDeviceCount(&deviceCount);
	for (int i = 0; i < deviceCount; i++) {
		hipDeviceProp_t props;
		hipGetDeviceProperties(&props, i);
		printf("Device nr.: %d\n", i);
		printf("  Device name: %s\n", props.name);
		printf("  Memory clock rate: (KHz) %d\n", props.memoryClockRate);
		printf("  Memory bus width (bits): %d\n", props.memoryBusWidth);
		printf("  Peak memory bandwith (GB/s): %f\n\n", 2.0*props.memoryClockRate*(props.memoryBusWidth/8)/1.0e6);
	}

	int n = 1 << 20;
	float *x, *y, *d_x, *d_y;
	x = (float*)malloc(n*sizeof(float));
	y = (float*)malloc(n*sizeof(float));

	hipMalloc(&d_x, n*sizeof(float));
	hipMalloc(&d_y, n*sizeof(float));

	for (int i = 0; i < n; ++i) {
		x[i] = 1.0f;
		y[i] = 2.0f;
	}

	hipMemcpy(d_x, x, n*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_y, y, n*sizeof(float), hipMemcpyHostToDevice);

	saxpy<<<(n+255)/256, 256>>>(n, 2.0f, d_x, d_y);

	hipMemcpy(y, d_y, n*sizeof(float), hipMemcpyDeviceToHost);

	float maxError = 0.0f;
	for (int i = 0; i < n; ++i) {
		maxError = max(maxError, abs(y[i]-4.0f));
	}
	printf("Max error: %f\n", maxError);

	hipFree(d_x);
	hipFree(d_y);
	free(x);
	free(y);
}